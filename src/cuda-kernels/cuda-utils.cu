#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

#define FILENAME(line) FILENAME_FOR_EXCEPTIONS_CUDA("src/cuda-kernels/cuda-utils.cu", line)

#include "awkward-cuda/cuda-utils.h"

dim3 threads(int64_t length) {
  if (length > 1024) {
    return dim3(1024);
  }
  return dim3(length);
}

dim3 blocks(int64_t length) {
  if (length > 1024) {
    return dim3(ceil((length) / 1024.0));
  }
  return dim3(1);
}

dim3 threads_2d(int64_t length_x, int64_t length_y) {
  if (length_x > 32 && length_y > 32) {
    return dim3(32, 32);
  } else if (length_x > 32 && length_y <= 32) {
    return dim3(32, length_y);
  } else if (length_x <= 32 && length_y > 32) {
    return dim3(length_x, 32);
  } else {
    return dim3(length_x, length_y);
  }
}

dim3 blocks_2d(int64_t length_x, int64_t length_y) {
  if (length_x > 32 && length_y > 32) {
    return dim3(ceil(length_x / 32.0), ceil(length_y / 32.0));
  } else if (length_x > 32 && length_y <= 32) {
    return dim3(ceil(length_x / 32.0), 1);
  } else if (length_x <= 32 && length_y > 32) {
    return dim3(1, ceil(length_y / 32.0));
  } else {
    return dim3(1, 1);
  }
}

ERROR post_kernel_checks(ERROR* kernel_err = nullptr) {
    ERROR err;
    if(kernel_err != nullptr) {
        err = *kernel_err;
    }
    else {
        hipError_t cuda_err = hipGetLastError();
        if (cuda_err != hipSuccess) {
          err = failure(
              hipGetErrorString(err), kSliceNone, kSliceNone, FILENAME(__LINE__));
        }
        err = success();
    }
    hipDeviceSynchronize();
    return err;
}

ERROR awkward_cuda_ptr_device_num(int64_t* num, void* ptr) {
  hipPointerAttribute_t att;
  hipError_t status = hipPointerGetAttributes(&att, ptr);
  if (status != hipError_t::hipSuccess) {
    return failure_pass_through(hipGetErrorString(status), kSliceNone, kSliceNone, FILENAME(__LINE__));
  }
  *num = att.device;
  return success();
}

ERROR awkward_cuda_ptr_device_name(char* name, void* ptr) {
  hipPointerAttribute_t att;
  hipError_t status = hipPointerGetAttributes(&att, ptr);
  if (status != hipError_t::hipSuccess) {
    return failure_pass_through(hipGetErrorString(status), kSliceNone, kSliceNone, FILENAME(__LINE__));
  }
  hipDeviceProp_t dev_prop;
  status = hipGetDeviceProperties(&dev_prop, att.device);
  if (status != hipError_t::hipSuccess) {
    return failure_pass_through(hipGetErrorString(status), kSliceNone, kSliceNone, FILENAME(__LINE__));
  }
  strcpy(name, dev_prop.name);
  return success();
}

ERROR awkward_cuda_host_to_device(
  void* to_ptr,
  void* from_ptr,
  int64_t bytelength) {
  hipError_t memcpy_stat = hipMemcpy(
    to_ptr, from_ptr, bytelength, hipMemcpyHostToDevice);
  if (memcpy_stat != hipError_t::hipSuccess) {
    return failure_pass_through(hipGetErrorString(memcpy_stat), kSliceNone, kSliceNone, FILENAME(__LINE__));
  }
  else {
    return success();
  }
}

ERROR awkward_cuda_device_to_host(
  void* to_ptr,
  void* from_ptr,
  int64_t bytelength) {
  hipError_t memcpy_stat = hipMemcpy(to_ptr,
                                       from_ptr,
                                       bytelength,
                                       hipMemcpyDeviceToHost);
  if (memcpy_stat != hipError_t::hipSuccess) {
    return failure_pass_through(hipGetErrorString(memcpy_stat), kSliceNone, kSliceNone, FILENAME(__LINE__));
  }
  else {
    return success();
  }
}
